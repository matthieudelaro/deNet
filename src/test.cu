
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "deNet is ON!\n";
const char STR_LENGTH = 13;

__global__ void deNet()
{
    printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
    int num_threads = STR_LENGTH;
    int num_blocks = 1;
    deNet<<<num_blocks,num_threads>>>();
    hipDeviceSynchronize();

    return 0;
}

